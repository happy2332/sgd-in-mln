
#include <hip/hip_runtime.h>
extern "C"
__global__ void initKernel(int *d_A, long size, int val)
{
  long idx = blockIdx.x * blockDim.x + threadIdx.x;

  if(idx < size)
    d_A[idx] = val;
}

extern "C"
__global__ void regularCompactKernel(int *d_A, int *d_B, long size, int intervalSize)
{
  long idx = blockIdx.x * blockDim.x + threadIdx.x;

  if(idx < size)
    d_B[idx] = d_A[idx * intervalSize];
}

extern "C"
__global__ void sumKernel(int *d_A, long size)
{
  extern __shared__ float A[];

  long idx = blockIdx.x * blockDim.x + threadIdx.x;

  if(idx < size)
    A[threadIdx.x] = d_A[idx];

  __syncthreads();

  int iters = ceil(log2((float)blockDim.x));
  int n = 1, m = 2;

  for(int i = 0; i < iters; i++)
    {
      if((threadIdx.x + n) < blockDim.x && (threadIdx.x & (m-1)) == 0)
	A[threadIdx.x] += A[threadIdx.x + n];
      n <<= 1;
      m <<= 1;
      __syncthreads();
    }

  if(threadIdx.x == 0)
    d_A[idx] = A[0];
}
