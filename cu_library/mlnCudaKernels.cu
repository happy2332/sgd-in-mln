
#include <hip/hip_runtime.h>
extern "C"
__global__ void initDbIndexKernel(int totalVars, int totalPreds, int *d_varDomainSizes,
                                  int *d_predBaseIdx, int *d_predVarMat, int *d_dbIndex,
                                  long totalGroundings)
{
  long idx = blockIdx.x * blockDim.x + threadIdx.x;

  if(idx < totalGroundings)
    {
      long baseDbIndex = idx * totalPreds;
      for(int i = 0; i < totalPreds; i++)
	  d_dbIndex[baseDbIndex + i] = d_predBaseIdx[i];

      long n = idx;
      for(int i = totalVars-1; i >= 0; i--)
	{
	  int domainSize = d_varDomainSizes[i];
	  long temp = n / domainSize;
	  int val = n - temp * domainSize;
	  n = temp;

	  int basePredVarMatIndex = i * totalPreds;
	  for(int j = 0; j < totalPreds; j++)
	      d_dbIndex[baseDbIndex + j] += d_predVarMat[basePredVarMatIndex + j] * val;
	}
    }
}

extern "C"
__global__ void evalClauseKernel(int *d_satArray, int **d_interpretation, int *dbIndex,
                                 int *d_predicates, int *d_valTrue, int totalPreds, long totalGroundings)
{
  long idx = blockIdx.x * blockDim.x + threadIdx.x;

  if(idx < totalGroundings && d_satArray[idx] == 1)
    {
      long baseDbIndex = idx * totalPreds;
      int sat = 0;
      for(int i = 0; i < totalPreds; i++)
	{
	  int predId = d_predicates[i];
	  long interpretationIdx = dbIndex[baseDbIndex + i];
	  sat = max(sat, d_interpretation[predId][interpretationIdx] == d_valTrue[i]);
	}

      d_satArray[idx] = sat;
    }
}
